
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/AbsCriterion.cu"
#else

void THNN_(AbsCriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *output,
           int64_t reduction)
{
  THCUNN_check_shape(state, input, target);
  THCUNN_assertSameGPU(state, 3, input, target, output);

  if (reduction == Reduction::None) {
    THCTensor_(resizeAs)(state, output, input);
    THC_pointwiseApply3<real, real, real>(state, input, target, output,
                        abs_updateOutput_no_reduce_functor<real>());
    return;
  }

  THCTensor_(resize1d)(state, output, 1);

  ptrdiff_t size = THCTensor_(nElement)(state, input);

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);

  thrust::device_ptr<real> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<real> target_data(THCTensor_(data)(state, target));
  accreal sum = thrust::inner_product(input_data, input_data+size, target_data, (accreal)0, thrust::plus<accreal>(), abs_functor<real, accreal>());

  if (reduction == Reduction::ElementwiseMean)
    sum /= size;

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);

  THCTensor_(set1d)(state, output, 0, ScalarConvert<accreal, real>::to(sum));
}

void THNN_(AbsCriterion_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           int64_t reduction)
{
  THCUNN_check_shape(state, input, target);
  THCUNN_assertSameGPU(state, 4, input, target, gradOutput, gradInput);

  THCTensor_(resizeAs)(state, gradInput, input);

  if (reduction == Reduction::None) {
    THCUNN_check_shape(state, gradOutput, input);
    THC_pointwiseApply3<real, real, real>(state, input, target, gradInput,
                        abs_updateGradInput_no_reduce_functor<real>());
    THCTensor_(cmul)(state, gradInput, gradInput, gradOutput);
    return;
  }

  THCUNN_check_dim_size(state, gradOutput, 1, 0, 1);

  ptrdiff_t size = THCTensor_(nElement)(state, input);
  real norm = ScalarConvert<double, real>::to(reduction == Reduction::ElementwiseMean ? 1./size : 1.);

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);

  thrust::device_ptr<real> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<real> target_data(THCTensor_(data)(state, target));
  thrust::device_ptr<real> gradInput_data(THCTensor_(data)(state, gradInput));

  thrust::transform(input_data, input_data+size, target_data, gradInput_data,
                    abs_updateGradInput_functor<real>(norm, THCTensor_(get1d)(state, gradOutput, 0)));

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);
}

#endif
